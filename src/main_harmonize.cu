#include "hip/hip_runtime.h"
#include "../harmonize.git/harmonize/cpp/harmonize.h"
using namespace util;

#include <fstream>
#include <iostream>
#include <map>
#include <sstream>
#include <stdio.h>
#include <string>
#include <vector>

#include "node.h"
#include "file_parser.h"
#include "node_graph.h"
#include "common.h"

struct BFSProgramOp {
  using Type = void (*)(Node* node, unsigned int current_depth, int previous);

  template <typename PROGRAM>
  __device__ static void eval(PROGRAM prog, Node *node, unsigned int current_depth, int previous) {
    // use as baseline for CPU version
    atomicCAS(&node->visited, 0, 1);
    atomicExch(&node->previous, previous);

    // Simpler version, without loop coalescing
    /*
    for (int i = 0; i < node->edge_count; i++) {

      unsigned int edge_node_id = prog.device.edge_arr[node->edge_offset + i];
      Node& edge_node = prog.device.node_arr[edge_node_id];

      if (atomicMin(&edge_node.depth, current_depth+1) > current_depth+1) {
        prog.template async<BFSProgramOp>(&edge_node, current_depth + 1, node);
      }
    }
    */

    //* with loop coalescing
    for (int i = 0; i < node->edge_count; i++) {
      int edge_node_id;
      bool hit = false;
      while ( (!hit) && (i < node->edge_count) ){
        edge_node_id = prog.device.edge_arr[node->edge_offset + i];
        Node& edge_node = prog.device.node_arr[edge_node_id];
        if (atomicMin(&edge_node.depth, current_depth+1) > current_depth+1) {
          hit = true;
          break;
        }
        i++;
      }
      if ( hit ) {
        Node& edge_node = prog.device.node_arr[edge_node_id];
        prog.template async<BFSProgramOp>(&edge_node, current_depth + 1, node->id);
      }
    }
    //*/

  }


};

// The device state, itself, is an immutable struct, but can contain references
// and pointers to non-const data.
struct MyDeviceState {
  Node* node_arr;
  int* edge_arr;
  int root_node;
  iter::AtomicIter<unsigned int>* iterator;
};

struct BFSProgramSpec {
  typedef OpUnion<BFSProgramOp> OpSet;
  typedef MyDeviceState DeviceState;

  static const size_t STASH_SIZE =   16;
  static const size_t FRAME_SIZE = 8191;
  static const size_t POOL_SIZE  = 8191;

  /*
    type PROGRAM {
      device: DeviceState
      template: Op
    }
  */

  // called by each work group at start
  template <typename PROGRAM> __device__ static void initialize(PROGRAM prog) {}

  // called by each work group at end
  template <typename PROGRAM> __device__ static void finalize(PROGRAM prog) {}

  // called by each work group if need work
  template <typename PROGRAM> __device__ static bool make_work(PROGRAM prog) {
    unsigned int index;

    if (prog.device.iterator->step(index)) {
      Node &root = prog.device.node_arr[prog.device.root_node];
      atomicMin(&root.depth,0);
      prog.template async<BFSProgramOp>(&root, 0, -1);
    }

    return false;
  }
};

using AsyncProgType = AsyncProgram<BFSProgramSpec>;
using EventProgType = EventProgram<BFSProgramSpec>;

template<typename ProgType, typename ProgTypeInstance>
void run_kernel(MyDeviceState ds, unsigned int arena_size, unsigned int group_count, unsigned int cycle_count) {
  ProgTypeInstance instance(arena_size, ds);
  hipDeviceSynchronize();
  host::check_error();

  // init program instance
  init<ProgType>(instance, 32);
  hipDeviceSynchronize();
  host::check_error();

  // exec program instance
  do {
    // Give the number of work groups and the size of the chunks pulled from the io buffer
    exec<ProgType>(instance, group_count, cycle_count);
    hipDeviceSynchronize();
    host::check_error();
  } while (!instance.complete());
}

int main_harmonize(int argc, char *argv[]) {
  cli::ArgSet args(argc, argv);
  bool directed = args["directed"];

  // arguments
  unsigned int group_count = args["group_count"] | args["group-count"]; // batch count
  unsigned int cycle_count = args["cycle_count"] | args["cycle-count"]; // run count
  unsigned int arena_size = args["arena_size"] | args["arena-size"] | 0x100000; // amount of memory to allocate

  char* file_str = args.get_flag_str((char*)"file");
  if (file_str == nullptr) {
    std::cerr << "no value provided for -file" << std::endl;
    std::exit(1);
  }

  char* program_type = args.get_flag_str((char*)"program");
  if (program_type == nullptr) {
    std::cerr << "no value provided for -program" << std::endl;
    std::exit(1);
  }

  // init DeviceState
  MyDeviceState ds;
  ds.root_node = args["root"]; // int

  iter::AtomicIter<unsigned int> host_iter(0, 1);
  host::DevBuf<iter::AtomicIter<unsigned int>> iterator;
  iterator << host_iter;
  ds.iterator = iterator;

  FileParser file_parser;
  if (file_parser.known_extension(file_str))
    file_parser.parse_extension(file_str);
  else
    file_parser.parse_arguments(args);

  std::map<int, std::vector<int>>& adjacency_graph = file_parser.parse_file(file_str, directed);
  NodeGraph node_graph(adjacency_graph);

  host::DevBuf<int> dev_edges(node_graph.edges.size());
  dev_edges << node_graph.edges;
  ds.edge_arr = dev_edges;

  host::DevBuf<Node> dev_nodes(node_graph.nodes.size());
  dev_nodes << node_graph.nodes;
  ds.node_arr = dev_nodes;

  Stopwatch watch;
  watch.start();

  if (std::string(program_type) == "async") {
    run_kernel<AsyncProgType, AsyncProgType::Instance>(ds, arena_size, group_count, cycle_count);
  } else if (std::string(program_type) == "event") {
    run_kernel<EventProgType, EventProgType::Instance>(ds, arena_size, group_count, cycle_count);
  }

  watch.stop();
  float msec = watch.ms_duration();

  std::vector<Node> out_host;
  dev_nodes >> out_host;
  common_output(args, msec, out_host, "harmonize");

  return 0;
}

#ifndef MAIN
int main(int argc, char *argv[]) {
  return main_harmonize(argc, argv);
}
#endif
